#include "hip/hip_runtime.h"
#include "pso.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <ctime>

#define FN_TO_OPTIMIZE(x, y) holderTable(x, y)

 __device__ double function(double x, double y)
{
    return pow((x - 3.14), 2) + pow((y - 2.72), 2) + sin(3*x + 1.41) + sin(4*y - 1.73);
}

__device__ double sphereFunction(double x, double y)
{
    return x*x + y*y;
}

__device__ double holderTable(double x, double y)
{
    return -abs( sin(x) * cos(y) * exp( abs(1.0 - (sqrt(x * x + y * y)/M_PI)) ) );
}

__device__ double sixHumpCamelFunction(double x, double y)
{
    return (4.0 - 2.1*x*x + pow(x, 4)/3.0)*x*x + x*y + (-4 + 4*y*y)*y*y;
}

// Uniform random function that uses cuda_uniform_double and returns a value between a range
__device__ double cu_urand(double lowerBound, double upperBound, hiprandState_t *state)
{
    return lowerBound + (upperBound - lowerBound) * hiprand_uniform_double(state);
}

__global__ void particle(PositionValue *result, int iterations, double velW, double cogAccel, double socAccel,
                         int numXS, int numYS, double lower, double upper, hiprandState_t *state, time_t seedVal)
{
    __shared__ PositionValue globalBest;
    __shared__ int curIteration;
    __shared__ xyPair s_low;
    __shared__ xyPair s_upp;

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    hiprand_init(seedVal, idx, 0, &state[idx]);
    
    if(threadIdx.x == 0)
    {
        xyPair interval = {(upper - lower)/numXS, (upper - lower)/numYS};
        s_upp.x = upper - interval.x * (numXS - 1 - blockIdx.x);
        s_low.x = lower + interval.x * blockIdx.x;
        s_upp.y = upper - interval.y * (numYS - 1 - blockIdx.y);
        s_low.y = lower + interval.y * blockIdx.y;
        curIteration = 0;
        globalBest = {cu_urand(s_low.x, s_upp.x, state + idx),
                      cu_urand(s_low.y, s_upp.y, state + idx),
                      0};
        globalBest.val = FN_TO_OPTIMIZE(globalBest.pos.x, globalBest.pos.y);
    }
    __syncthreads();

    PositionValue curr = {cu_urand(s_low.x, s_upp.x, state + idx),
                          cu_urand(s_low.y, s_upp.y, state + idx),
                          0};
    curr.val = FN_TO_OPTIMIZE(curr.pos.x, curr.pos.y);
    
    PositionValue localBest = curr;

    xyPair velocity = {cu_urand(0, 2, state + idx), cu_urand(0, 2, state + idx)};

    while(curIteration < iterations)
    {
        if(localBest.val > curr.val)
        {
            localBest = curr;
            if(globalBest.val > localBest.val)
            {
                globalBest = localBest;
            }
        }
        __syncthreads();

        velocity.x = velW * velocity.x + cogAccel * cu_urand(0, 1, state + idx) * (localBest.x - curr.x) + socAccel * cu_urand(0, 1, state + idx) * (globalBest.x - curr.x);
        velocity.y = velW * velocity.y + cogAccel * cu_urand(0, 1, state + idx) * (localBest.y - curr.y) + socAccel * cu_urand(0, 1, state + idx) * (globalBest.y - curr.y);

        curr.pos.x += velocity.x;
        curr.pos.y += velocity.y;

        if(curr.pos.x < s_low.x)
            curr.pos.x = s_low.x;
        else if(curr.pos.x > s_upp.x)
            curr.pos.x = s_upp.x;

        if(curr.pos.y < s_low.y)
            curr.pos.y = s_low.y;
        else if(curr.pos.y > s_upp.y)
            curr.pos.y = s_upp.y;

        curr.val = FN_TO_OPTIMIZE(curr.pos.x, curr.pos.y);

        if(threadIdx.x == 0)
        {
            curIteration++;
        }
    }

    __syncthreads();
    if(threadIdx.x == 0)
    {
        result[0] = globalBest;
    }
}

void multiSwarmOptimizer(PositionValue *result, unsigned int numXS, unsigned int numYS,
                         int numP, int iterations, double velW, double cogAccel, double socAccel,
                         double lower, double upper)
{
    PositionValue *device_results;
    hiprandState_t *device_state;
    gpuErrchk( hipMalloc(&device_state, sizeof(hiprandState_t)) );
    gpuErrchk( hipMalloc(&device_results, sizeof(PositionValue) * numXS * numYS) );

    particle<<<{numXS, numYS}, numP>>>(device_results, iterations, velW, cogAccel, socAccel, numXS, numYS, lower, upper, device_state, time(0));

    gpuErrchk( hipMemcpy(result, device_results, sizeof(PositionValue) * numXS * numYS, hipMemcpyDeviceToHost) );

    gpuErrchk( hipFree(device_results) );
    gpuErrchk( hipFree(device_state) );
}
