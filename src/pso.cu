#include "hip/hip_runtime.h"
#include "pso.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <random>
#include <ctime>

// Macro to quickly change the function to be optimized
#define FN_TO_OPTIMIZE(x, y) holderTable(x, y)

// ===== List of all the Functions Used to Optimize =====
 __device__ double function(double x, double y)
{
    return pow((x - 3.14), 2) + pow((y - 2.72), 2) + sin(3*x + 1.41) + sin(4*y - 1.73);
}

__device__ double sphereFunction(double x, double y)
{
    return x*x + y*y;
}

__device__ double holderTable(double x, double y)
{
    return -abs( sin(x) * cos(y) * exp( abs(1.0 - (sqrt(x * x + y * y)/M_PI)) ) );
}

__device__ double sixHumpCamelFunction(double x, double y)
{
    return (4.0 - 2.1*x*x + pow(x, 4)/3.0)*x*x + x*y + (-4 + 4*y*y)*y*y;
}

__device__ double ackleyFunction(double x, double y)
{
    return -20.0 * exp( -0.2 * sqrt( (1.0/2.0) * (x*x + y*y) ) ) - exp( (1.0/2.0) * (cos(2*M_PI*x) + cos(2*M_PI*y)) ) + 20.0 + exp(1.0);
}
// ===== End of all the Functions Used to Optimize =====

// Uniform random function that uses cuda_uniform_double and returns a value between a range
__device__ double cu_urand(double lowerBound, double upperBound, hiprandState_t *state)
{
    return lowerBound + (upperBound - lowerBound) * hiprand_uniform_double(state);
}

// The actual kernel/particle function
__global__ void particle(PositionValue *results, int iterations, double velW,
                         double cogAccel, double socAccel, int numXS, int numYS,
                         double lower, double upper, hiprandState_t *state, time_t seedVal)
{
    // All the values used by all particles of the swarm
    __shared__ PositionValue globalBest;
    __shared__ int curIteration;
    __shared__ xyPair s_low; // Lower boundry of the swarm
    __shared__ xyPair s_upp; // Upper boundry of the swarm

    int id = blockIdx.y * gridDim.x + blockIdx.x + threadIdx.x;
    hiprand_init(seedVal, id, 0, &state[id]);
    
    // Initializes the shared memory used by all particles in a sub-swarm
    if(threadIdx.x == 0)
    {
        // Sets up the boundry used by all particles in the block
        xyPair interval = {(upper - lower)/numXS, (upper - lower)/numYS};
        s_upp.x = upper - interval.x * (numXS - 1 - blockIdx.x);
        s_low.x = lower + interval.x * blockIdx.x;
        s_upp.y = upper - interval.y * (numYS - 1 - blockIdx.y);
        s_low.y = lower + interval.y * blockIdx.y;

        curIteration = 0;

        // Randomly select a global best location within the boundries
        globalBest = {cu_urand(s_low.x, s_upp.x, state + id),
                      cu_urand(s_low.y, s_upp.y, state + id),
                      0};
        globalBest.val = FN_TO_OPTIMIZE(globalBest.pos.x, globalBest.pos.y);
    }

    __syncthreads();

    // Each particle spwans in a random location in the function within their boundries
    PositionValue curr = {cu_urand(s_low.x, s_upp.x, state + id),
                          cu_urand(s_low.y, s_upp.y, state + id),
                          0};
    curr.val = FN_TO_OPTIMIZE(curr.pos.x, curr.pos.y);
    
    PositionValue localBest = curr; // The first location is their best location as of right now

    xyPair velocity = {cu_urand(0, 2, state + id), cu_urand(0, 2, state + id)}; // Set up a random velocity value

    // The core iteration loop
    while(curIteration < iterations)
    {
        // Update best values if a new minimum is found
        if(localBest.val > curr.val)
        {
            localBest = curr;
            if(globalBest.val > localBest.val)
            {
                globalBest = localBest;
            }
        }
        __syncthreads();

        // Update the velocity of the particle using the PSO velocity function
        // v_new = w * vel_old + c1 * r1 * (personalBest - curr) + c2 * r2 * (globalBest - curr)
        // where w is weight, c1 is cogAccel, c2 is socialAccel, and r1, r2 are random numbers from 0 to 1
        velocity.x = velW * velocity.x + cogAccel * cu_urand(0, 1, state + id) * (localBest.pos.x - curr.pos.x) + socAccel * cu_urand(0, 1, state + id) * (globalBest.pos.x - curr.pos.x);
        velocity.y = velW * velocity.y + cogAccel * cu_urand(0, 1, state + id) * (localBest.pos.y - curr.pos.y) + socAccel * cu_urand(0, 1, state + id) * (globalBest.pos.y - curr.pos.y);

        // Change the position of the particle based on the updated velocities
        curr.pos.x += velocity.x;
        curr.pos.y += velocity.y;

        // Prevent the particle from going out of their boundry
        if(curr.pos.x < s_low.x)
            curr.pos.x = s_low.x;
        else if(curr.pos.x > s_upp.x)
            curr.pos.x = s_upp.x;

        if(curr.pos.y < s_low.y)
            curr.pos.y = s_low.y;
        else if(curr.pos.y > s_upp.y)
            curr.pos.y = s_upp.y;

        curr.val = FN_TO_OPTIMIZE(curr.pos.x, curr.pos.y);

        // Only one thread needs to change the current iteration
        if(threadIdx.x == 0)
        {
            curIteration++;
        }
    }

    __syncthreads();

    // Write out the best result obtained by the swarm
    if(threadIdx.x == 0)
    {
        results[id] = globalBest;
    }
}

void multiSwarmOptimizer(PositionValue *results, unsigned int numXS, unsigned int numYS,
                         int numP, int iterations, double velW, double cogAccel, double socAccel,
                         double lower, double upper)
{
    PositionValue *device_results;
    hiprandState_t *device_state; // Used for the random number generator
    
    gpuErrchk( hipMalloc(&device_state, sizeof(hiprandState_t)) );
    gpuErrchk( hipMalloc(&device_results, sizeof(PositionValue) * numXS * numYS) );
    
    // The creation of the multi-swarm universe
    particle<<<{numXS, numYS}, numP>>>(device_results, iterations, velW, cogAccel, socAccel,
                                       numXS, numYS, lower, upper, device_state, time(0));

    gpuErrchk( hipDeviceSynchronize() ); 
    gpuErrchk( hipMemcpy(results, device_results, sizeof(PositionValue) * numXS * numYS, hipMemcpyDeviceToHost) );

    gpuErrchk( hipFree(device_results) );
    gpuErrchk( hipFree(device_state) );
}
